#include <stdio.h>
int main()
{
    // Get device properties
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %zu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %zu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %zu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %zu\n",  devProp.totalConstMem);
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Texture alignment:             %zu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));    return 0;
    printf("Integrated:      %u\n",  (devProp.integrated));
    printf("Can Host Memory:      %u\n",  (devProp.canMapHostMemory));
    printf("Compute Mode:      %u\n",  (devProp.computeMode));
    printf("Concurrent Kernels:      %u\n",  (devProp.concurrentKernels));
    printf("ECCEnabled:      %u\n",  (devProp.ECCEnabled));
    printf("pciBusID:      %u\n",  (devProp.pciBusID));
    printf("pciDeviceID:      %u\n",  (devProp.pciDeviceID));
    printf("tccDriver:      %u\n",  (devProp.tccDriver));
}