#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <time.h> 
#include <math.h>
#include <sys\timeb.h>
#include <ctime>

#define PI 3.1415926
using namespace std;

__host__ void outer_compute (int *arr1, int *arr2, int* out, float* sum1, float* sum2, int N, int blockSize);
__global__ void addKernel(int* d_arr1, int* d_arr2, int* d_out, float* d_sum1, float* d_sum2, int N, int blockSize);
__device__ int calculate(int val1, int val2);


__device__ int calculate(int val1, int val2)
{
	return val1*val2;
}

__global__ void addKernel(int* d_arr1, int* d_arr2, int* d_dot_out, float* d_sum1, float* d_sum2, int N, int blockSize)
{
	int i;
	d_dot_out[threadIdx.x] = 0;
	d_sum1[threadIdx.x] = 0;
	d_sum2[threadIdx.x] = 0;
	for(i = 0; i <= N/blockSize && (i*blockSize + threadIdx.x) < N; i++)
	{
		int val1 = d_arr1[i*blockSize + threadIdx.x];
		int val2 = d_arr2[i*blockSize + threadIdx.x];
		d_dot_out[threadIdx.x] += calculate(val1, val2);
		d_sum1[threadIdx.x] += (val1*val1);
		d_sum2[threadIdx.x] += (val2*val2);
	}
}


__host__ void outer_compute (int *arr1, int *arr2, int* dot_out, float* sum1, float* sum2, int N, int blockSize) {
	int *d_arr1, *d_arr2, *d_dot_out;
	float *d_sum1, *d_sum2;
	hipMalloc((void **) &d_arr1, N*sizeof(int));
	hipMalloc((void **) &d_arr2, N*sizeof(int));	

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	hipMemcpy(d_arr1, arr1, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_arr2, arr2, N*sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(stop);
    hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time for the Host to Device transfer : %f ms\n", milliseconds);

	hipMalloc((void **) &d_dot_out, blockSize*sizeof(int));
	hipMalloc((void **) &d_sum1, blockSize*sizeof(float));
	hipMalloc((void **) &d_sum2, blockSize*sizeof(float));

	dim3 dimGrid(1, 1);
	dim3 dimBlock(blockSize);

	//Compute
	hipEventRecord(start);
	addKernel<<<dimGrid,dimBlock>>> (d_arr1, d_arr2, d_dot_out, d_sum1, d_sum2, N, blockSize);
	hipDeviceSynchronize();
	hipEventRecord(stop);
    hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time for the kernel execution : %f ms\n", milliseconds);
	
	hipEventRecord(start);
	hipMemcpy(dot_out, d_dot_out, blockSize*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(sum1, d_sum1, blockSize*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(sum2, d_sum2, blockSize*sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop);
    hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time for the Device to Host transfer : %f ms\n", milliseconds);
	hipFree(d_arr1); 
	hipFree(d_arr2); 
	hipFree(d_dot_out);
	hipFree(d_sum1);
	hipFree(d_sum2);
}

int main(int argc, char *argv[])
{
	srand(time(0)); 
	int N, blockSize;
    N = atoi(argv[1]);
    blockSize = atoi(argv[2]);

    int* arr1 = (int*)malloc(N * sizeof(int));
	int* arr2 = (int*)malloc(N * sizeof(int));
	int* dot_out = (int*)malloc(blockSize * sizeof(int));
	float* sum1 = (float*)malloc(blockSize * sizeof(float));
	float* sum2 = (float*)malloc(blockSize * sizeof(float));

    printf("\n\nInfo\n-----------------\n");
    printf("Number of Elements: %d\n", N);
    printf("Number of threads per block: %d\n", blockSize);
    printf("Number of blocks will be created: %d\n", N/blockSize);

	printf("Time\n-----------------\n");
	clock_t begin = clock();
	if(argv[3] == NULL){
		for(int i = 0; i < N; i++){
	        arr1[i] = rand()%100;
	        arr2[i] = rand()%100; 
	    }
	}else{
	    char* filename = argv[3];
	    int i = 0;
	    char line[10];	 
	    FILE* fp = fopen(filename, "r");
	    if (fp == NULL){
	        printf("Could not open file %s",filename);
	        return 0;
	    }
	    while (fgets(line, sizeof(line), fp)){
	    	if(i <= N && i > 0){
	    		arr1[i-1]= atoi(line);
	    	}
	    	else if(i > N){
	    		arr2[(i-N)-1]= atoi(line);
	    	}
	        i++;
	    }
	    fclose(fp);
	}


    clock_t end = clock();
  	double elapsed_secs = double(end - begin);
  	printf("Time for the array generation : %f ms\n", elapsed_secs);

	begin = clock();
	int cpu_dot_total = 0;
	float cpu_vector_a = 0;
	float cpu_vector_b = 0;
	float cpu_sum = 0;
	for(int i = 0; i < N; i++){
		cpu_dot_total += arr1[i] * arr2[i];
		cpu_vector_a += arr1[i]*arr1[i];
		cpu_vector_b += arr2[i]*arr2[i];
	}
	cpu_vector_a = sqrt(cpu_vector_a);
	cpu_vector_b = sqrt(cpu_vector_b);
	cpu_sum = cpu_vector_a*cpu_vector_b;
    float cpu_angle = acos(cpu_dot_total/cpu_sum)*180/PI;
    end = clock();
  	elapsed_secs = double(end - begin);
  	printf("Time for the CPU function : %f ms\n", elapsed_secs);

    /*
    for(int i = 0; i < N; i++){
    	if(i%blockSize == 0){
    		printf("Here is the 0\n");
    	}
    	printf("Arr1 : %d\n", arr1[i]);
    }

    for(int i = 0; i < N; i++){
    	if(i%blockSize == 0){
    		printf("Here is the 0\n");
    	}
    	printf("Arr2 : %d\n", arr2[i]);
    }
*/

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
    outer_compute(arr1, arr2, dot_out, sum1, sum2, N, blockSize);
    int dot_total = 0;
    float sum1_total = 0;
    float sum2_total = 0;
    float sum = 0;
    float angle = 0;
    for(int i = 0; i < blockSize; i++){
    	//printf("Hop1: %d\n", dot_out[i]);
    	dot_total += dot_out[i];
    	//printf("Hop2: %f\n", sum1[i]);
    	sum1_total += sum1[i];
    	//printf("Hop3: %f\n", sum2[i]);
    	sum2_total += sum2[i];
    }
    sum1_total = sqrt(sum1_total);
    sum2_total = sqrt(sum2_total);
    sum = sum1_total*sum2_total;
    angle = acos(dot_total/sum)*180/PI;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Total execution time for GPU : %f ms\n", milliseconds);
    printf("Results\n-----------------\n");
    printf("The CPU angle is: %.3f\n", cpu_angle);
    printf("The GPU angle is: %.3f", angle);
    return 0;
}

